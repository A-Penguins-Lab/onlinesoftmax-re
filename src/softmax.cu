#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <utility>
#include <math.h>

using namespace std;

// Define the threadDim and blockDim from here
constexpr int THREADDIM = 128;

__global__ void softmax(float* vecA, float* sum, int N) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if (idx < N) {
		vecA[idx] = vecA[idx] / *sum;  // dereferencing sum properly
	}
}

__global__ void ExponentiateKernel(float* vecA, int N) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	
	if (idx < N) {
		vecA[idx] = expf(vecA[idx]);
	}
}

__global__ void SumReduction(float* vecA, float* sum, int N) {
    __shared__ float sdata[THREADDIM];
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int tid = threadIdx.x;

    sdata[tid] = (idx < N) ? vecA[idx] : 0.0f;
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
      *sum = sdata[0];
    }
}

__global__ void maxReduction(float* vecA, float* max, int N){ 
	unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
	float maxVal;

	for (unsigned int stride = blockDim.x; stride >= 1; stride /= 2){
		if (threadIdx.x < stride) {
			if (vecA[idx + stride] > vecA[idx]) {
				maxVal = vecA[idx + stride];
			}
			__syncthreads();
	
		}
		__syncthreads();

	}
	
	if (threadIdx.x == 0) {
		*max = maxVal;
	}
}

float* randomVec(int N) {
	float* M = new float[N];
	
	for (int i = 0; i < N; i++) {
		M[i] = 0.21f * i;
	}

  return M;
}

void RunSoftMax(int N) {
  float* M = randomVec(N);
  float *Md, *sumd;

	hipMalloc(&Md, sizeof(float) * N);
  hipMalloc(&sumd, sizeof(float)); 

  hipMemset(sumd, 0, sizeof(float));  // Important: initialize sum to 0
	hipMemcpy(Md, M, sizeof(float) * N, hipMemcpyHostToDevice);

  int threadsPerBlock = THREADDIM;
  int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

	// Launch kernels
	ExponentiateKernel<<<blocksPerGrid, threadsPerBlock>>>(Md, N);
    SumReduction<<<blocksPerGrid, threadsPerBlock>>>(Md, sumd, N);


	float sum;
	hipMemcpy(&sum, sumd, sizeof(float), hipMemcpyDeviceToHost);

  softmax<<<blocksPerGrid, threadsPerBlock>>>(Md, sumd, N);

  float* result = new float[N];
  hipMemcpy(result, Md, sizeof(float) * N, hipMemcpyDeviceToHost);

  // Debug print
  for (int i = 0; i < N; i++) {
		cout << result[i] << " ";
  }
  cout << endl;

	// Clean up
	hipFree(Md);
	hipFree(sumd);

	delete[] M;
	delete[] result;

	cout << "Softmax done and so are all allocs and frees" << endl;
}

int main() {
  int N = 128;
	RunSoftMax(N);
	return 0;
}
